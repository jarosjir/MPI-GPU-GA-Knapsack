/*
 * File:        Parameters.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *
 *              and
 *
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 *
 * Comments:    The implementation  parameter class. It contains all the parameters of
 *              GA and knapsack
 *
 *
 * License:     This source code is distribute under OpenSource GNU GPL license
 *
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php
 *
 *
 *
 * Created on 08 June     2012, 00:00 PM
 * Revised on 24 February 2022, 16:24 PM
 */


#include <iostream>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <getopt.h>

#include "Parameters.h"
#include "CUDAKernels.h"

//----------------------------------------------------------------------------//
//                              Definitions                                   //
//----------------------------------------------------------------------------//
extern __constant__  TEvolutionParameters GPU_EvolutionParameters;

// Singleton initialization
bool TParameters::pTParametersInstanceFlag = false;
TParameters* TParameters::pTParametersSingle = NULL;


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              public methods                                //
//----------------------------------------------------------------------------//

/*
 * Get instance of TPrarams
 */
TParameters* TParameters::GetInstance(){
    if(! pTParametersInstanceFlag)
    {
        pTParametersSingle = new TParameters();
        pTParametersInstanceFlag = true;
        return pTParametersSingle;
    }
    else
    {
        return pTParametersSingle;
    }
}// end of TParameters::GetInstance
//-----------------------------------------------------------------------------


/*
 * Load parameters from command line
 *
 * @param argc
 * @param argv
 *
 */
void TParameters::LoadParametersFromCommandLine(int argc, char **argv){


   float OffspringPercentage = 0.5f;
   float EmigrantPercentage = 0.1f;
   char c;



   while ((c = getopt (argc, argv, "p:g:m:c:o:e:n:f:s:bh")) != -1){
       switch (c){
          case 'p':{
              if (atoi(optarg) != 0) EvolutionParameters.PopulationSize = atoi(optarg);
              break;
          }
          case 'g': {
              if (atoi(optarg) != 0) EvolutionParameters.NumOfGenerations = atoi(optarg);
              break;
          }


          case 'm': {
              if (atof(optarg) != 0) EvolutionParameters.MutationPst = atof(optarg);
              break;
          }
          case 'c': {
              if (atof(optarg) != 0) EvolutionParameters.CrossoverPst = atof(optarg);
              break;
          }
          case 'o': {
              if (atof(optarg) != 0) OffspringPercentage = atof(optarg);;
              break;
          }


         case 'e': {
              if (atof(optarg) != 0) EmigrantPercentage = atof(optarg);;
              break;
          }
          case 'n': {
              if (atoi(optarg) != 0) EvolutionParameters.MigrationInterval = atoi(optarg);
              break;
          }

         case 's': {
              if (atoi(optarg) != 0) EvolutionParameters.StatisticsInterval = atoi(optarg);
              break;
          }

         case 'b': {
              FPrintBest = true;
              break;
          }

         case 'f': {
              GlobalDataFileName  = optarg;
              break;
          }
          case 'h':{

             PrintUsageAndExit();
             break;
          }
          default:{

               PrintUsageAndExit();
          }
       }
   }

   // Set population size to be even.
   if (EvolutionParameters.PopulationSize % 2 == 1) EvolutionParameters.PopulationSize++;

   EvolutionParameters.OffspringPopulationSize = (int) (OffspringPercentage * EvolutionParameters.PopulationSize);
   if (EvolutionParameters.OffspringPopulationSize == 0) EvolutionParameters.OffspringPopulationSize = 2;
   if (EvolutionParameters.OffspringPopulationSize % 2 == 1) EvolutionParameters.OffspringPopulationSize++;

     // Check emigrant count and set it at least to 1
   EvolutionParameters.EmigrantCount = (int) (EmigrantPercentage * EvolutionParameters.PopulationSize);
   if (EvolutionParameters.EmigrantCount == 0)  EvolutionParameters.EmigrantCount = 1;
   if ((EvolutionParameters.EmigrantCount % 2) == 0) EvolutionParameters.EmigrantCount++;

   if (EvolutionParameters.EmigrantCount > EvolutionParameters.PopulationSize)  EvolutionParameters.EmigrantCount = EvolutionParameters.PopulationSize;

   if (EvolutionParameters.MigrationInterval < 0) EvolutionParameters.MigrationInterval = 1;

    // Set UINT mutation threshold to faster comparison
   EvolutionParameters.MutationUINTBoundary  = (unsigned int) ((float) UINT_MAX * EvolutionParameters.MutationPst);
   EvolutionParameters.CrossoverUINTBoundary = (unsigned int) ((float) UINT_MAX * EvolutionParameters.CrossoverPst);


   // Set island Idx and Island count
   MPI_Comm_rank(MPI_COMM_WORLD, &EvolutionParameters.IslandIdx);
   MPI_Comm_size(MPI_COMM_WORLD, &EvolutionParameters.IslandCount);

   SetGPU();

} // end of LoadParametersFromCommandLine
//------------------------------------------------------------------------------


/*
 * Copy parameters to the GPU constant memory
 */
void TParameters::StoreParamsOnGPU()
{
  hipMemcpyToSymbol(HIP_SYMBOL(GPU_EvolutionParameters), &EvolutionParameters, sizeof(TEvolutionParameters));
  checkAndReportCudaError(__FILE__,__LINE__);
}// end of StoreParamsOnGPU
//------------------------------------------------------------------------------


/*
 * Return GPU id attached to the MPI process
 */
void TParameters::SetGPU()
{
    // Get number of devices per node (must be uniform accross nodes!)
    int nDevices = -1;
    hipGetDeviceCount(&nDevices);
    checkAndReportCudaError(__FILE__,__LINE__);

    // MPI processes are consecutive on the node. All nodes have to be equipped
    // with the same number of GPUs
    FGPUIdx = EvolutionParameters.IslandIdx % nDevices;

    hipSetDevice(FGPUIdx);
    checkAndReportCudaError(__FILE__,__LINE__);

    hipDeviceProp_t 	prop;
    hipGetDeviceProperties (&prop, FGPUIdx);
    checkAndReportCudaError(__FILE__,__LINE__);

    FGPU_SM_Count = prop.multiProcessorCount;

}// end of GetGPUIdx
//------------------------------------------------------------------------------


//----------------------------------------------------------------------------//
//                              Implementation                                //
//                              private methods                               //
//----------------------------------------------------------------------------//

/*
 * Constructor of the class
 */
TParameters::TParameters(){

    EvolutionParameters.PopulationSize      = 128;
    EvolutionParameters.ChromosomeSize      = 128;
    EvolutionParameters.NumOfGenerations    = 100;

    EvolutionParameters.MutationPst         = 0.01f;
    EvolutionParameters.CrossoverPst        = 0.7f;
    EvolutionParameters.OffspringPopulationSize = (int) (0.5f * EvolutionParameters.PopulationSize);

    EvolutionParameters.IslandCount         = 1;
    EvolutionParameters.EmigrantCount       = 1;
    EvolutionParameters.MigrationInterval   = 1;
    EvolutionParameters.StatisticsInterval  = 1;

    EvolutionParameters.IntBlockSize        = sizeof(int)*8;
    GlobalDataFileName                      = "";

    FPrintBest                              = false;
    EvolutionParameters.IslandIdx           = 0;

}// end of TParameters
//------------------------------------------------------------------------------

/*
 * print usage of the algorithm
 */
void TParameters::PrintUsageAndExit(){

  if (EvolutionParameters.IslandIdx == 0){
      cerr << "Usage: " << endl;
      cerr << "  -p Population_size\n";
      cerr << "  -g Number_of_generations\n";
      cerr << endl;

      cerr << "  -m mutation_rate\n";
      cerr << "  -c crossover_rate\n";
      cerr << "  -o offspring_rate\n";
      cerr << endl;

      cerr << "  -e emigrants_rate\n";
      cerr << "  -n migration_interval\n";
      cerr << "  -s statistics_interval\n";
      cerr << endl;

      cerr << "  -b print best individual\n";
      cerr << "  -f benchmark_file_name\n";


      cerr << endl;
      cerr << "Default Population_size       = 128"  << endl;
      cerr << "Default Number_of_generations = 100" << endl;
      cerr << endl;

      cerr << "Default mutation_rate  = 0.01" << endl;
      cerr << "Default crossover_rate = 0.7" << endl;
      cerr << "Default offspring_rate = 0.5" << endl;
      cerr << endl;

      cerr << "Default island_count        = 1"   << endl;
      cerr << "Default emigrants_rate      = 0.1" << endl;
      cerr << "Default migration_interval  = 1"   << endl;
      cerr << "Default statistics_interval = 1"   << endl;

      cerr << "Default benchmark_file_name = knapsack_data.txt\n";

  }
  MPI_Finalize();
  exit(1);

}// end of PrintUsage
//------------------------------------------------------------------------------





/*
 * Print all parameters
 *
 */
void TParameters::PrintAllParameters(){

    if (EvolutionParameters.IslandIdx == 0){
        printf("-----------------------------------------\n");
        printf("--- Evolution parameters --- \n");
        printf("Population size:     %d\n", EvolutionParameters.PopulationSize);
        printf("Offspring size:      %d\n", EvolutionParameters.OffspringPopulationSize);
        printf("Chromosome int size: %d\n", EvolutionParameters.ChromosomeSize);
        printf("Chromosome size:     %d\n", EvolutionParameters.ChromosomeSize * EvolutionParameters.IntBlockSize);

        printf("Num of generations:  %d\n", EvolutionParameters.NumOfGenerations);
        printf("\n");


        printf("Crossover pst:       %f\n", EvolutionParameters.CrossoverPst);
        printf("Mutation  pst:       %f\n", EvolutionParameters.MutationPst);
        printf("Crossover int:       %u\n", EvolutionParameters.CrossoverUINTBoundary);
        printf("Mutation  int:       %u\n", EvolutionParameters.MutationUINTBoundary);
        printf("\n");

        printf("Emigrant count:      %d\n", EvolutionParameters.EmigrantCount);
        printf("Migration interval:  %d\n", EvolutionParameters.MigrationInterval);
        printf("Island count:        %d\n", EvolutionParameters.IslandCount);
        printf("Statistics interval: %d\n", EvolutionParameters.StatisticsInterval);

        printf("\n");
        printf("Data File: %s\n",GlobalDataFileName.c_str());
        printf("-----------------------------------------\n");
    }

}// end of PrintAllParameters
//------------------------------------------------------------------------------
