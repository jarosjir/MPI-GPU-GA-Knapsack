/**
 * @file        Parameters.cpp
 * @author      Jiri Jaros
 *              Brno University of Technology
 *              Faculty of Information Technology
 *
 *              and
 *
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 *              jarosjir@fit.vutbr.cz
 *              www.fit.vutbr.cz/~jarosjir
 *
 * @brief       Header file of the parameter class.
 *              This class maintains all the parameters of evolution.
 *
 * @date        30 March     2012, 00:00 (created)
 *              25 February  2022, 20:01 (revised)
 *
 * @copyright   Copyright (C) 2012 - 2022 Jiri Jaros.
 *
 * This source code is distribute under OpenSouce GNU GPL license.
 * If using this code, please consider citation of related papers
 * at http://www.fit.vutbr.cz/~jarosjir/pubs.php
 *
 */


#include <mpi.h>
#include <getopt.h>
#include <hip/hip_runtime_api.h>

#include "Parameters.h"
#include "CUDAKernels.h"

//--------------------------------------------------------------------------------------------------------------------//
//--------------------------------------------------- Definitions ----------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/// Copy of Evolutionary parameters in device constant memory.
extern __constant__  EvolutionParameters gpuEvolutionParameters;

// Singleton initialization
bool Parameters::sInstanceFlag             = false;
Parameters* Parameters::sSingletonInstance = nullptr;


//--------------------------------------------------------------------------------------------------------------------//
//------------------------------------------------- Public methods ---------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Get instance of Parameters
 */
Parameters& Parameters::getInstance()
{
  if(!sInstanceFlag)
  {
    sSingletonInstance = new Parameters();
    sInstanceFlag = true;
    return *sSingletonInstance;
  }
  else
  {
    return *sSingletonInstance;
  }
}// end of Parameters::getInstance
//----------------------------------------------------------------------------------------------------------------------

/**
 * Load parameters from command line
 */
void Parameters::parseCommandline(int    argc,
                                  char** argv)
{
  float offspringPercentage = 0.5f;
  float emigrantPercentage = 0.1f;
  char c;

  while ((c = getopt (argc, argv, "p:g:m:c:o:e:n:f:s:bh")) != -1)
  {
    switch (c)
    {
      case 'p':
      {
        if (atoi(optarg) != 0)
        {
          mEvolutionParameters.populationSize = atoi(optarg);
        }
        break;
      }
      case 'g':
      {
        if (atoi(optarg) != 0)
        {
          mEvolutionParameters.numOfGenerations = atoi(optarg);
        }
        break;
      }
      case 'm':
      {
        if (atof(optarg) != 0)
        {
          mEvolutionParameters.mutationPst = atof(optarg);
        }
        break;
      }
     case 'c':
     {
        if (atof(optarg) != 0)
        {
          mEvolutionParameters.crossoverPst = atof(optarg);
        }
        break;
      }
      case 'o':
      {
        if (atof(optarg) != 0)
        {
          offspringPercentage = atof(optarg);
        }
        break;
      }
      case 'e':
      {
        if (atof(optarg) != 0)
        {
          emigrantPercentage = atof(optarg);
        }
        break;
      }
      case 'n':
      {
        if (atoi(optarg) != 0)
        {
          mEvolutionParameters.migrationInterval = atoi(optarg);
        }
        break;
      }
      case 's':
      {
        if (atoi(optarg) != 0)
        {
          mEvolutionParameters.statisticsInterval = atoi(optarg);
        }
        break;
      }
      case 'b':
      {
        mPrintBest = true;
        break;
      }
      case 'f':
      {
        mGlobalDataFileName  = optarg;
        break;
      }
      case 'h':
      {
        printUsageAndExit();
        break;
      }
      default:
      {
        printUsageAndExit();
      }
    }
  }

  // Set population size to be even.
  if (mEvolutionParameters.populationSize % 2 == 1)
  {
    mEvolutionParameters.populationSize++;
  }

  mEvolutionParameters.offspringPopulationSize = (int) (offspringPercentage * mEvolutionParameters.populationSize);
  if (mEvolutionParameters.offspringPopulationSize == 0)
  {
    mEvolutionParameters.offspringPopulationSize = 2;
  }
  if (mEvolutionParameters.offspringPopulationSize % 2 == 1)
  {
    mEvolutionParameters.offspringPopulationSize++;
  }

  // Check emigrant count and set it at least to 1
  mEvolutionParameters.emigrantCount = (int) (emigrantPercentage * mEvolutionParameters.populationSize);
  if (mEvolutionParameters.emigrantCount == 0)
  {
    mEvolutionParameters.emigrantCount = 1;
  }
  if ((mEvolutionParameters.emigrantCount % 2) == 0)
  {
    mEvolutionParameters.emigrantCount++;
  }

  if (mEvolutionParameters.emigrantCount > mEvolutionParameters.populationSize)
  {
    mEvolutionParameters.emigrantCount = mEvolutionParameters.populationSize;
  }

  if (mEvolutionParameters.migrationInterval < 0)
  {
    mEvolutionParameters.migrationInterval = 1;
  }

  // Set UINT mutation threshold to faster comparison
  mEvolutionParameters.mutationUintBoundary  = (unsigned int) ((float) UINT_MAX * mEvolutionParameters.mutationPst);
  mEvolutionParameters.crossoverUintBoundary = (unsigned int) ((float) UINT_MAX * mEvolutionParameters.crossoverPst);

  // Set island Idx and Island count
  MPI_Comm_rank(MPI_COMM_WORLD, &mEvolutionParameters.islandIdx);
  MPI_Comm_size(MPI_COMM_WORLD, &mEvolutionParameters.islandCount);

  setDevice();
} // end of parseCommandline
//----------------------------------------------------------------------------------------------------------------------

/**
 * Copy parameters to the GPU constant memory.
 */
void Parameters::copyToDevice()
{
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(gpuEvolutionParameters), &mEvolutionParameters, sizeof(mEvolutionParameters)));
}// end of copyToDevice
//----------------------------------------------------------------------------------------------------------------------

/**
 * Set device idx attached to the MPI process.
 */
void Parameters::setDevice()
{
  // Get number of devices per node (must be uniform across nodes!)
  int nDevices = -1;
  checkCudaErrors(hipGetDeviceCount(&nDevices));

  // MPI processes are consecutive on the node. All nodes have to be equipped
  // with the same number of GPUs
  mDeviceIdx = mEvolutionParameters.islandIdx % nDevices;

  checkCudaErrors(hipSetDevice(mDeviceIdx));


  hipDeviceProp_t 	prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, mDeviceIdx));


  mNumberOfDeviceSM = prop.multiProcessorCount;
}// end of setDevice
//----------------------------------------------------------------------------------------------------------------------

/**
 * Print all parameters.
 */
void Parameters::printOutAllParameters()
{
  if (mEvolutionParameters.islandIdx == 0)
  {
    printf("-----------------------------------------\n");
    printf("--- Evolution parameters --- \n");
    printf("Population size:     %d\n", mEvolutionParameters.populationSize);
    printf("Offspring size:      %d\n", mEvolutionParameters.offspringPopulationSize);
    printf("Chromosome int size: %d\n", mEvolutionParameters.chromosomeSize);
    printf("Chromosome size:     %d\n", mEvolutionParameters.chromosomeSize * mEvolutionParameters.intBlockSize);

    printf("Num of generations:  %d\n", mEvolutionParameters.numOfGenerations);
    printf("\n");


    printf("Crossover pst:       %f\n", mEvolutionParameters.crossoverPst);
    printf("Mutation  pst:       %f\n", mEvolutionParameters.mutationPst);
    printf("Crossover int:       %u\n", mEvolutionParameters.crossoverUintBoundary);
    printf("Mutation  int:       %u\n", mEvolutionParameters.mutationUintBoundary);
    printf("\n");

    printf("Emigrant count:      %d\n", mEvolutionParameters.emigrantCount);
    printf("Migration interval:  %d\n", mEvolutionParameters.migrationInterval);
    printf("Island count:        %d\n", mEvolutionParameters.islandCount);
    printf("Statistics interval: %d\n", mEvolutionParameters.statisticsInterval);

    printf("\n");
    printf("Data File: %s\n",mGlobalDataFileName.c_str());
    printf("-----------------------------------------\n");
  }
}// end of printOutAllParameters
//----------------------------------------------------------------------------------------------------------------------


//--------------------------------------------------------------------------------------------------------------------//
//------------------------------------------------ Private methods ---------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Constructor of the class
 */
Parameters::Parameters()
{
  mEvolutionParameters.populationSize      = 128;
  mEvolutionParameters.chromosomeSize      = 128;
  mEvolutionParameters.numOfGenerations    = 100;

  mEvolutionParameters.mutationPst         = 0.01f;
  mEvolutionParameters.crossoverPst        = 0.7f;
  mEvolutionParameters.offspringPopulationSize = (int) (0.5f * mEvolutionParameters.populationSize);

  mEvolutionParameters.islandCount         = 1;
  mEvolutionParameters.emigrantCount       = 1;
  mEvolutionParameters.migrationInterval   = 1;
  mEvolutionParameters.statisticsInterval  = 1;

  mEvolutionParameters.intBlockSize        = sizeof(int) * 8;
  mGlobalDataFileName                      = "";

  mPrintBest                              = false;
  mEvolutionParameters.islandIdx           = 0;

}// end of Parameters
//----------------------------------------------------------------------------------------------------------------------

/**
 * print usage of the algorithm
 */
void Parameters::printUsageAndExit()
{
  if (mEvolutionParameters.islandIdx == 0)
  {
    fprintf(stderr, "Parameters for the genetic algorithm solving knapsack problem: \n");
    fprintf(stderr, "  -p population_size\n");
    fprintf(stderr, "  -g number_of_generations\n");
    fprintf(stderr, "\n");

    fprintf(stderr, "  -m mutation_rate\n");
    fprintf(stderr, "  -c crossover_rate\n");
    fprintf(stderr, "  -o offspring_rate\n");
    fprintf(stderr, "\n");

    fprintf(stderr, "  -e emigrants_rate\n");
    fprintf(stderr, "  -n migration_interval\n");
    fprintf(stderr, "  -s statistics_interval\n");

    fprintf(stderr, "  -b print best individual\n");
    fprintf(stderr, "  -f benchmark_file_name\n");

    fprintf(stderr, "\n");
    fprintf(stderr, "Default population_size       = 128\n");
    fprintf(stderr, "Default number_of_generations = 100\n");
    fprintf(stderr, "\n");

    fprintf(stderr, "Default mutation_rate  = 0.01\n");
    fprintf(stderr, "Default crossover_rate = 0.7\n");
    fprintf(stderr, "Default offspring_rate = 0.5\n");
    fprintf(stderr, "\n");

    fprintf(stderr, "Default island_count        = 1\n");
    fprintf(stderr, "Default migration_interval  = 1\n");
    fprintf(stderr, "Default statistics_interval = 1\n");

    fprintf(stderr, "Default benchmark_file_name = knapsack_data.txt\n");
  }

  MPI_Finalize();
  exit(EXIT_FAILURE);
}// end of printUsageAndExit
//----------------------------------------------------------------------------------------------------------------------
