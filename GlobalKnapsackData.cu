#include "hip/hip_runtime.h"
/*
 * File:        GlobalKnapsackData.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *
 *              and
 *
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 *
 * Comments:    Implementation file of the knapsack global data class.
 *              This class maintains the benchmark data
 *
 *
 * License:     This source code is distribute under OpenSource GNU GPL license
 *
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php
 *
 *
 *
 * Created on 08 June     2012, 00:00 PM
 * Revised on 24 February 2022, 18:59 PM
 */


#include <hip/hip_runtime_api.h>
#include <fstream>

#include "GlobalKnapsackData.h"
#include "Parameters.h"

//--------------------------------------------------------------------------------------------------------------------//
//--------------------------------------------------- Definitions ----------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

static const char* const ERROR_FILE_NOT_FOUND = "Global Benchmark Data: File not found\n";


//--------------------------------------------------------------------------------------------------------------------//
//------------------------------------------------- Public methods ---------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Destructor of the class
 */
GlobalKnapsackData::~GlobalKnapsackData()
{
  freeMemory();
}// end of GlobalKnapsackData
//----------------------------------------------------------------------------------------------------------------------

/**
 * Load data from file, filename given in Parameter class.
 */
void GlobalKnapsackData::loadFromFile()
{
  // Get instance of Parameter class
  Parameters& params = Parameters::getInstance();

  // Open file with benchmark data
  std::ifstream fr(params.getBenchmarkFileName().c_str());
  if (!fr.is_open())
  {
    fprintf(stderr, ERROR_FILE_NOT_FOUND);
    exit(EXIT_FAILURE);
  }

  // Read number of items
  int numberOfItems = 0;
  fr >> numberOfItems;

  const int originalNumberOfItems = numberOfItems;

  // Calculate padding
  int overhead = numberOfItems % (params.getIntBlockSize() * WARP_SIZE);
  if (overhead != 0)
  {
    numberOfItems = numberOfItems + ((params.getIntBlockSize() * WARP_SIZE) - overhead);
  }

  // Allocate memory for arrays
  allocateMemory(numberOfItems);

  mHostData->numberOfItems         = numberOfItems;
  mHostData->originalNumberOfItems = originalNumberOfItems;


  // Load prices
  for (size_t i = 0; i < originalNumberOfItems; i++)
  {
    fr >> mHostData->itemPrice[i];
  }
  // add padding
  for (size_t i = originalNumberOfItems; i < numberOfItems; i++)
  {
    mHostData->itemPrice[i] = PriceType(0);
  }


  // Load weights
  for (size_t i = 0; i < originalNumberOfItems; i++)
  {
    fr >> mHostData->itemWeight[i];
  } // add padding
  for (size_t i = originalNumberOfItems; i < numberOfItems; i++)
  {
    mHostData->itemWeight[i] = PriceType(0);
  }

  // Get max Price/Weight ratio
  mHostData->maxPriceWightRatio = 0.0f;

  for (size_t i = 0; i < originalNumberOfItems; i++)
  {
    if (mHostData->itemWeight[i] != 0)
    {
      float ratio = mHostData->itemPrice[i] / mHostData->itemWeight[i];
      if (ratio > mHostData->maxPriceWightRatio)
      {
        mHostData->maxPriceWightRatio = ratio;
      }
    }
  }

  // Read Knapsack capacity
  fr >> mHostData->knapsackCapacity;

  // Update chromosome size in parameters
  params.setChromosomeSize(numberOfItems / params.getIntBlockSize());


  // Upload global data to device memory
  copyToDevice();
}// end of loadFromFile
//----------------------------------------------------------------------------------------------------------------------

//--------------------------------------------------------------------------------------------------------------------//
//----------------------------------------------- Protected methods --------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Allocate memory
 */
void GlobalKnapsackData::allocateMemory(int numberOfItems)
{
  //------------------------- Host allocation ------------------------------//
  checkCudaErrors(
      hipHostAlloc<KnapsackData>(&mHostData,  sizeof(KnapsackData), hipHostMallocDefault)
  );

  checkCudaErrors(
      hipHostAlloc<PriceType>(&mHostData->itemPrice,  sizeof(PriceType) * numberOfItems, hipHostMallocDefault)
  );

  checkCudaErrors(
      hipHostAlloc<WeightType>(&mHostData->itemWeight,  sizeof(WeightType) * numberOfItems, hipHostMallocDefault)
  );


  //----------------------- Device allocation ------------------------------//
  checkCudaErrors(
      hipMalloc<KnapsackData>(&mDeviceData,  sizeof(KnapsackData) )
  );

  checkCudaErrors(
      hipMalloc<PriceType>(&mDeviceItemPriceHandler,  sizeof(PriceType) * numberOfItems)
  );

  checkCudaErrors(
      hipMalloc<WeightType>(&mDeviceItemWeightHandler, sizeof(WeightType) * numberOfItems)
  );
}// end of allocateMemory
//----------------------------------------------------------------------------------------------------------------------

/**
 * Free Memory.
 */
void GlobalKnapsackData::freeMemory()
{

    //------------------------- Host allocation ------------------------------//
    checkCudaErrors(hipHostFree(mHostData->itemPrice));
    checkCudaErrors(hipHostFree(mHostData->itemWeight));
    checkCudaErrors(hipHostFree(mHostData));

    //----------------------- Device allocation ------------------------------//
    checkCudaErrors(hipFree(mDeviceData));
    checkCudaErrors(hipFree(mDeviceItemPriceHandler));
    checkCudaErrors(hipFree(mDeviceItemWeightHandler));

}// end of freeMemory
//----------------------------------------------------------------------------------------------------------------------

/**
 * Upload Data to Device.
 */
void GlobalKnapsackData::copyToDevice()
{
  // Copy basic structure - struct data
  checkCudaErrors(hipMemcpy(mDeviceData, mHostData, sizeof(KnapsackData), hipMemcpyHostToDevice));


  // Set pointer of the ItemPrice vector into the struct on GPU (link struct and vector)
  checkCudaErrors(
      hipMemcpy(&(mDeviceData->itemPrice), &mDeviceItemPriceHandler, sizeof(PriceType*),hipMemcpyHostToDevice)
  );


    // Set pointer of the ItemWeight vector into struct on GPU (link struct and vector)
  checkCudaErrors(
      hipMemcpy(&(mDeviceData->itemWeight), &mDeviceItemWeightHandler, sizeof(WeightType*), hipMemcpyHostToDevice)
  );

  // Copy prices
  checkCudaErrors(
      hipMemcpy(mDeviceItemPriceHandler, mHostData->itemPrice,  sizeof(PriceType) * mHostData->numberOfItems,
                    hipMemcpyHostToDevice)
  );

  // Copy weights
  checkCudaErrors(
      hipMemcpy(mDeviceItemWeightHandler, mHostData->itemWeight, sizeof(WeightType) * mHostData->numberOfItems,
                 hipMemcpyHostToDevice)
  );

}// end of copyToDevice
//----------------------------------------------------------------------------------------------------------------------