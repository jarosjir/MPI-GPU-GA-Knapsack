#include "hip/hip_runtime.h"
/*
 * File:        GlobalKnapsackData.cu
 * Author:      Jiri Jaros
 * Affiliation: Brno University of Technology
 *              Faculty of Information Technology
 *
 *              and
 *
 *              The Australian National University
 *              ANU College of Engineering & Computer Science
 *
 * Email:       jarosjir@fit.vutbr.cz
 * Web:         www.fit.vutbr.cz/~jarosjir
 *
 * Comments:    Implementation file of the knapsack global data class.
 *              This class maintains the benchmark data
 *
 *
 * License:     This source code is distribute under OpenSource GNU GPL license
 *
 *              If using this code, please consider citation of related papers
 *              at http://www.fit.vutbr.cz/~jarosjir/pubs.php
 *
 *
 *
 * Created on 08 June     2012, 00:00 PM
 * Revised on 24 February 2022, 16:25 PM
 */


#include <fstream>
#include <iostream>

#include "GlobalKnapsackData.h"
#include "Parameters.h"


//----------------------------------------------------------------------------//
//                              Definitions                                   //
//----------------------------------------------------------------------------//

static const char * ERROR_FILE_NOT_FOUND = "Global Benchmark Data: File not found";


//----------------------------------------------------------------------------//
//                              public methods                                //
//----------------------------------------------------------------------------//

/*
 * Constructor of the class
 */
TGlobalKnapsackData::TGlobalKnapsackData(){
    DeviceData = NULL;
    HostData   = NULL;

    FDeviceItemPriceHandler  = NULL;
    FDeviceItemWeightHandler = NULL;

}// end of constructor
//------------------------------------------------------------------------------


/*
 * Destructor of the class
 */
TGlobalKnapsackData::~TGlobalKnapsackData(){

    FreeMemory();

}// end of TGlobalKnapsackData
//------------------------------------------------------------------------------



/*
 * Load data from file, filename given in Parameter class
 */
void TGlobalKnapsackData::LoadFromFile(){


    // Get instance of Parameter class
    TParameters * Params = TParameters::GetInstance();

    // Open file with benchmark data
    ifstream fr(Params->BenchmarkFileName().c_str());

    if (!fr.is_open()) {
        cerr << ERROR_FILE_NOT_FOUND << endl;
        Params->PrintUsageAndExit();
    }


    // Read number of items
    int NumberOfItems = 0;
    fr >>NumberOfItems;

    int OriginalNumberOfItems = NumberOfItems;

    // Calculate padding
    int Overhead = NumberOfItems % (Params->IntBlockSize() * WARP_SIZE);
    if (Overhead != 0) NumberOfItems = NumberOfItems + ((Params->IntBlockSize() * WARP_SIZE) - Overhead);


    // Allocate memory for arrays
    AllocateMemory(NumberOfItems);

    HostData->NumberOfItems         = NumberOfItems;
    HostData->OriginalNumberOfItems = OriginalNumberOfItems;



    //-- load price --//
    for (size_t i = 0; i < OriginalNumberOfItems; i++){
        fr >> HostData->ItemPrice[i];

    }
    for (size_t i = OriginalNumberOfItems; i < NumberOfItems; i++){
        HostData->ItemPrice[i] = TPriceType(0);
    }



    //-- load weight --//
    for (size_t i = 0; i < OriginalNumberOfItems; i++){
        fr >> HostData->ItemWeight[i];
    }

    for (size_t i = OriginalNumberOfItems; i < NumberOfItems; i++){
        HostData->ItemWeight[i] = TPriceType(0);
    }


    //-- get max ratio --//
    HostData->MaxPriceWightRatio = 0.0f;

    for (size_t i = 0; i < OriginalNumberOfItems; i++){
        if (HostData->ItemWeight[i] != 0) {
                float Ratio = HostData->ItemPrice[i] / HostData->ItemWeight[i];
                if (Ratio > HostData->MaxPriceWightRatio)  HostData->MaxPriceWightRatio = Ratio;
        }

    }


    //Read Knapsack capacity
    fr >> HostData->KnapsackCapacity;

    // Update chromosome size in parameters
    Params->SetChromosomeSize(NumberOfItems/Params->IntBlockSize());

    // Upload global data to device memory
    UploadDataToDevice();

}// end of LoadFromFile
//------------------------------------------------------------------------------



//----------------------------------------------------------------------------//
//                           protected methods                                //
//----------------------------------------------------------------------------//

/*
 * Allocate memory
 *
 * @param       NumberOfItems - Number of Items in Knapsack with padding
 */
void TGlobalKnapsackData::AllocateMemory(int NumberOfItems){


    //------------------------- Host allocation ------------------------------//
    //------------------- All data allocated by PINNED memory ----------------//

    hipHostAlloc((void**)&HostData,  sizeof(TKnapsackData), hipHostMallocDefault);

    hipHostAlloc((void**)&HostData->ItemPrice,  sizeof(TPriceType) * NumberOfItems, hipHostMallocDefault);

    hipHostAlloc((void**)&HostData->ItemWeight,  sizeof(TWeightType)* NumberOfItems, hipHostMallocDefault);




    //----------------------- Device allocation ------------------------------//

    hipMalloc((void**)&(DeviceData),  sizeof(TKnapsackData) );

    hipMalloc((void**)&(FDeviceItemPriceHandler),  sizeof(TPriceType) * NumberOfItems);

    hipMalloc((void**)&(FDeviceItemWeightHandler),  sizeof(TWeightType) * NumberOfItems);




}// end of AllocateMemory
//------------------------------------------------------------------------------


/*
 * Free Memory
 */
void TGlobalKnapsackData::FreeMemory(){

    if (HostData) {

        //------------------------- Host free --------------------------------//
        if (HostData->ItemPrice)
          hipHostFree(HostData->ItemPrice);


        if (HostData->ItemWeight)
          hipHostFree(HostData->ItemWeight);


       hipHostFree(HostData);

    }



    //----------------------- Device free ------------------------------------//
    if (DeviceData)

       hipFree(DeviceData);



    if (FDeviceItemPriceHandler)

       hipFree(FDeviceItemPriceHandler);


    if (FDeviceItemWeightHandler)

       hipFree(FDeviceItemWeightHandler);




}// end of AllocateMemory
//------------------------------------------------------------------------------



/*
 * Upload Data to Device
 */
void TGlobalKnapsackData::UploadDataToDevice(){




    // Copy basic structure - struct data

         hipMemcpy(DeviceData, HostData, sizeof(TKnapsackData),
                    hipMemcpyHostToDevice);


    // Set pointer of the ItemPrice vector into the struct on GPU (link struct and vector)

         hipMemcpy(&(DeviceData->ItemPrice), &FDeviceItemPriceHandler, sizeof(TPriceType * ),
                    hipMemcpyHostToDevice);



    // Set pointer of the ItemWeight vector into struct on GPU (link struct and vector)

         hipMemcpy(&(DeviceData->ItemWeight), &FDeviceItemWeightHandler, sizeof(TWeightType * ),
                    hipMemcpyHostToDevice);




    // Copy prices

         hipMemcpy(FDeviceItemPriceHandler, HostData->ItemPrice,  sizeof(TPriceType) * HostData->NumberOfItems,
                    hipMemcpyHostToDevice);


    // Copy weights

         hipMemcpy(FDeviceItemWeightHandler, HostData->ItemWeight, sizeof(TWeightType) * HostData->NumberOfItems,
                    hipMemcpyHostToDevice);



}// end of UploadDataToDevice
//------------------------------------------------------------------------------
